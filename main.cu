#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include <boost/math/special_functions/beta.hpp>
#include <boost/math/special_functions/gamma.hpp>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <math.h>
//#include <math_functions.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <sys/timeb.h>
#include <time.h>
#include <vector>

#include "definitions.cuh"


// Error handling for CUDA errors
static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}


void printHelpMessage() {
  printf("Required arguments:\n"
         "  -d input.txt\n"
         "  -c classfile.txt\n"
         "  -g geneset.txt (GMT format)\n\n"
         "Optional arguments:\n"
         "  -mp {integer} (# of parents)\n"
         "  -rs {integer} (number of resamplings, default = 100)\n"
         "  -prs {float} (resampling proportions, default = 0.9)\n"
         "  -r {integer} (number of permutations, defaul = 100)\n"
         "  -pw {float} (prior weight = [0, 1], default = 0.5)\n"
         "  -rawp (if set, no multiple testing; Bonferroni correction if not set.)\n"
         "  -pE {float} (p-value threshold for edge signficance, default = 0.05)\n"
         "  -pD {float} (p-value threshold for DDN, default = 0.05)\n"
         "  -randseed {integer} (a random seed; default = 0.\n"
         "                       if set to -1, time will be used.)\n\n"
         );
}

int main(int argc, char *argv[]) {
  // looking at GPU properties
  int nDevices;

  // int maxBlocks;
  int maxThreads;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Processor Clock Rate (KHz): %d\n", prop.clockRate);
    printf("  Device Max Number of Blocks: %d\n", prop.maxGridSize[1]);
    printf("  Device Max Number of Threads per Block: %d\n",
           prop.maxThreadsPerBlock);

    maxThreads = prop.maxThreadsPerBlock;
    printf("  Device Max Number of Compute Indices: %d\n",
           prop.maxGridSize[1] * prop.maxThreadsPerBlock);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    printf("  Compute Capability : %d.%d\n", prop.major, prop.minor);
    printf("  Device has %d SMs\n", prop.multiProcessorCount);
    printf("  This device can run multiple kernels simultaneously : %d \n\n",
           prop.concurrentKernels);
  }

  int startT = getMilliCount();
  int start1 = getMilliCount();

  // data grab routine
  // *************************************************************************************
  // command line arguments
  // parser--------------------------------------------------------------------------

  // Random Seed
  int random_seed = 0;  // default see;  if set to -1, random seed will be set to time.

  // gene expression ternary or binary valued data
  // the first column is gene id, and the first line is sample ids
  char *inputFile = NULL;

  // class names in a single line with tab delimited
  char *classFile = NULL;

  // gene set file in GMT format
  char *genesetFile = NULL;

  // limits the number of parents a node can have
  int parentCap = 0;

  // number of resampling
  int n_resamplings = 100;
  double prop_resampling = 0.9;

  // number of permutations, to estimate alpha and beta, for beta distribution
  int perms = 100; // 100 permutations

  // prior weight: 0 = no prior weight, 0.5 (default), and 1.0 = full prior
  // weight
  double priorWeight = 0.5;

  double alphaDDN =
      0.05; // default value (alpha) for statistical significance of DDN

  // to be deprecated
  double theta = -1;
  double lambda = 2.0;

  // [deprecated]
  // these are thresholds converted from theta and lambda
  // double thresh;
  // double threshpw; // threshold with priorWeight

  // new thresholds for significance of edges, either with no prior or with
  // prior
  double alphaEdge = 0.05,
         alphaEdgePrior = 1 - (1 - alphaEdge) * pow(priorWeight, 1 / lambda);

  // flag for multiple testing correction
  bool flag_pAdjust = true;

  //-d for input
  //-g for geneset
  //-c for class
  //-mp for max parents
  //-pD for p threshold value for DDN significance
  //-rs number of resamplings
  //-prs resampling proportion
  //-r number of permutations
  //-pE for p threshold value for edge signficance
  //-rawp (no additional value afterward), given no multiple testing correction
  //-pw prior weight = [0,1]
  //-randseed random seed
  //-t for theta (to be deprecated)
  //-l for lambda (to be deprecated)
  // loop through argv, determining location of each arg parameter

  if (argc < 2) {
    printHelpMessage();
    exit(EXIT_FAILURE);
  }

  for (int i = 1; i < argc; i++) {
    if (strcmp(argv[i], "-d") == 0)
      inputFile = argv[i + 1];
    else if (strcmp(argv[i], "-g") == 0)
      genesetFile = argv[i + 1];
    else if (strcmp(argv[i], "-c") == 0)
      classFile = argv[i + 1];
    else if (strcmp(argv[i], "-l") == 0)
      lambda = atof(argv[i + 1]);
    else if (strcmp(argv[i], "-t") == 0)
      theta = atof(argv[i + 1]);
    else if (strcmp(argv[i], "-mp") == 0)
      parentCap = atoi(argv[i + 1]);
    else if (strcmp(argv[i], "-rs") == 0)
      n_resamplings = atoi(argv[i + 1]);
    else if (strcmp(argv[i], "-prs") == 0)
      prop_resampling = atof(argv[i + 1]);
    else if (strcmp(argv[i], "-r") == 0)
      perms = atoi(argv[i + 1]);
    else if (strcmp(argv[i], "-pw") == 0 || strcmp(argv[i], "pW") == 0)
      priorWeight = atof(argv[i + 1]);
    else if (strcmp(argv[i], "-pE") == 0)
      alphaEdge = atof(argv[i + 1]);
    else if (strcmp(argv[i], "-rawp") == 0)
      flag_pAdjust = false;
    else if (strcmp(argv[i], "-pD") == 0)
      alphaDDN = atof(argv[i + 1]);
    else if (strcmp(argv[i], "-randseed") == 0)
      random_seed = atoi(argv[i + 1]);
    else if (strcmp(argv[i], "-help") == 0 || strcmp(argv[i], "--help") == 0)
      printHelpMessage();
  }

  // set to defaults if no arguments are included
  if (inputFile == NULL) {
    printf("Invalid input file entered. Exiting...\n");
    exit(EXIT_FAILURE);
  }

  if (classFile == NULL) {
    printf("Invalid class file entered. Exiting...\n");
    exit(EXIT_FAILURE);
  }

  if (genesetFile == NULL) {
    printf("Invalid geneset file entered. Exiting...\n");
    exit(EXIT_FAILURE);
  }
  printf("%s\n", genesetFile);

  if (alphaDDN < 0.0 || alphaDDN > 1.0) {
    alphaDDN = .05; // default if out of range
  }
  printf("alpha for signficance of DDN  = %f\n", alphaDDN);

  // set maxparents to default 3 if not set in command line arguments
  if (parentCap <= 0) {
    // should normally be run with 3 which will make it for a total of 4
    parentCap = 3;
  }
  printf("Max parents = %d\n", parentCap);

  const int MAX_PARENTS = parentCap;
  if (perms <= 0) {
    perms = 100; // default
  }
  printf("Permutations = %d\n", perms);

  if (priorWeight < 0.0 || priorWeight > 1.0) {
    priorWeight = 0.5; // half and half prior knowledge
  }
  printf("priorWeight : %f\n", priorWeight);

  // if -t is given,
  if (theta > 0) {
    alphaEdge = 1 - pow(theta, 1.0 / lambda);
  } else {
    theta = pow(1 - alphaEdge, lambda);
  }

  // alpha for edge detection, with and without prior
  alphaEdgePrior = 1 - (1 - alphaEdge) * pow(priorWeight, 1 / lambda);
  printf("alphaEdge : %f, alphaEdgePrior : %f\n", alphaEdge, alphaEdgePrior);

  // multiple testing correction
  printf("Multiple testing correction (edge detection) : %s\n", flag_pAdjust ? "true" : "false");

  // multiple testing correction
  printf("Random seed : %d\n", random_seed);

  // to be deprecated in the future
  printf("[to be deprecated] lambda : %f, theta : %f\n", lambda, theta);
  // end command line
  // parser---------------------------------------------------------------------------------------

  // expression data
  FILE *fpExpr = fopen(inputFile, "r");

  // class
  FILE *fpClass = fopen(classFile, "r");

  // gene set
  FILE *fpGeneSet = fopen(genesetFile, "r");

  FILE *fpResults = fopen("results.txt", "w");

  // check that files are working
  if (fpExpr == NULL) {
    printf("Expression file is NULL: %s [%d]\n", strerror(errno), errno);
    exit(EXIT_FAILURE);
  }
  if (fpClass == NULL) {
    printf("Class File File is NULL: %s [%d]\n", strerror(errno), errno);
    exit(EXIT_FAILURE);
  }
  if (fpGeneSet == NULL) {
    printf("Gene List File is NULL: %s [%d]\n", strerror(errno), errno);
    exit(EXIT_FAILURE);
  }
  if (fpResults == NULL) {
    printf("results file is NULL: %s [%d]\n", strerror(errno), errno);
    exit(EXIT_FAILURE);
  }

  // allocate memory for file reads
  //char buf[MAX_BUFSIZE];
  char *buf, *buf2,ch;
  char sampleNames[MAX_NUM_SAMPLES][MAX_LENGTH_NAME];
  char classNames[MAX_NUM_SAMPLES][MAX_LENGTH_NAME];
  char genesetGenes[MAX_GENESET_SIZE][MAX_LENGTH_NAME];
  int classIDs[MAX_NUM_SAMPLES];
  char class1[MAX_LENGTH_NAME], class2[MAX_LENGTH_NAME];
  char geneNames[MAX_NUM_GENES][MAX_LENGTH_NAME];
  int genesetIndexIntoData[MAX_GENESET_SIZE];

  int firstlinecharcount=0;
  int numSamples = 0;
  int numGenes = 0, genesetLength = 0;
  int numClass1, numClass2;

  int i, j;
  int *data;
  int *transferData1;
  int *transferData2;
  char *token;

  short *resampled_indices_C1;  // to keep track of resampling (C1)
  unsigned int n_resampled_C1;

  short *resampled_indices_C2;  // to keep track of resampling (C2)
  unsigned int n_resampled_C2;

  // determine length of first line
  while ((ch=getc(fpExpr)) != '\n') {                                             
      firstlinecharcount++;                                                       
  } 

  fseek(fpExpr, 0, 0);

  buf = (char *)malloc(firstlinecharcount * sizeof(char));

  // loads expression file into buffer
  // THIS LINE CAUSES PROBLEMS
  fgets(buf, sizeof(buf)*firstlinecharcount, fpExpr);
  token = strtok(buf, "\t");
  token = strtok(NULL, "\t");

  // skip first colum, gene_id
  // load samplenames from buffer and count number of samples
  while (token != NULL) {
    strcpy(sampleNames[numSamples], token);
    numSamples++;
    token = strtok(NULL, "\t");
  }

  for (int i = 0; i < numSamples; i++) {
    // Get rid of extra empty "sample" caused by trailing tab
    if (strlen(sampleNames[i]) == 1)
      numSamples--;
  }

  printf("%d samples\n", numSamples);

  while (fgets(buf, firstlinecharcount, fpExpr)) {
    token = strtok(buf, "\t");
    strcpy(geneNames[numGenes], token);
    //    printf("%s\n", genenames[numgenes]);
    numGenes++;
  }

  numGenes--;
  printf("%d genes\n", numGenes);

  data = (int *)malloc(numGenes * numSamples * sizeof(int));
  // reset file position to 0
  fseek(fpExpr, 0, 0);
  // Skip first line
  fgets(buf, firstlinecharcount, fpExpr);
  fgets(buf, firstlinecharcount, fpExpr);
  for (i = 0; i < numGenes; i++) {
    fgets(buf, firstlinecharcount, fpExpr);
    token = strtok(buf, "\t");
    for (j = 0; j < numSamples; j++) {
      token = strtok(NULL, "\t");
      assert(i * numSamples + j < numGenes * numSamples);
      sscanf(token, "%d", &data[i * numSamples + j]);
    }
  }

  fclose(fpExpr);
  firstlinecharcount=0;
  while ((ch=getc(fpClass)) != '\n') {                                             
      firstlinecharcount++;                                                       
  } 
  fseek(fpClass, 0, 0); 

  buf2 = (char *)malloc((firstlinecharcount+1) * sizeof(char));
  fgets(buf2, firstlinecharcount+1, fpClass);
  token = strtok(buf2, "\t");
  for (i = 0; i < numSamples; i++) {
    strcpy(classNames[i], token);
    token = strtok(NULL, "\t");
  }
  for (i = 1; i < numSamples; i++) {
    if (strcmp(classNames[i], classNames[0]))
      break;
  }
  strcpy(class1, classNames[0]);
  strcpy(class2, classNames[i]);
  numClass1 = 0;
  numClass2 = 0;

  // check if classfile had newline character at end of final classname,
  // preventing strcmp from working
  if (classNames[numSamples - 1][strlen(classNames[numSamples - 1]) - 1] ==
      '\n') {
    classNames[numSamples - 1][strlen(classNames[numSamples - 1]) - 1] = '\0';
  }

  for (i = 0; i < numSamples; i++) {
    if (!strcmp(classNames[i], class1)) {
      numClass1++;
      classIDs[i] = 0;
    }
    if (!strcmp(classNames[i], class2)) {
      numClass2++;
      classIDs[i] = 1;
    }
  }

  // should this be done? when reading in CTRP sample data not all classids are
  // filled this leads to a problem when reshuffling them later - only finding
  // 201/202 supposed samples
  /*int revisedSamples = 0;
     for (int i = 0; i < numsamples; i++)
     {
     if (classids[i] == 1 || classids[i] == 0)
     {
     revisedSamples++;
     }
     }
     printf("revised : %d original : %d\n", revisedSamples, numsamples);
     numsamples = revisedSamples; */

  //  printf("\n");
  printf("Classes: %d %s, %d %s\n", numClass1, class1, numClass2, class2);
  fclose(fpClass);
  clock_t cpuTime = clock(), diff;
  //-----------------------------------------------------------------------------------
  //-----------------------------------------------------------------------------------

  // fprintf(results, "%s\t%s\t%s\t\n", class2, "JS", "P");
  fprintf(fpResults, "Pathway\tJS\tPval\tnGenes\n");
  while (fgets(buf, firstlinecharcount, fpGeneSet)) {
    // check if beginning of file is newline
    if (buf[0] == '\n') {
      continue;
    }
    // ensures that classids are restored to a preshuffled state
    // for the first run of each pathway before permutations begin
    for (int k = 0; k < numSamples; k++) {
      if (strcmp(classNames[k], class1) == 0) {
        classIDs[k] = 0;
      }
      if (strcmp(classNames[k], class2) == 0) {
        classIDs[k] = 1;
      }
    }

    char *pathwayName = strtok(buf, "\n");
    // fgets(buf, sizeof(buf), fp3);
    // token = strtok(buf, "\t");
    token = strtok(pathwayName, "\t");
    token = strtok(NULL, "\t");
    // Get first word "Geneset"
    token = strtok(NULL, "\t");
    // Skip second word "URL"
    genesetLength = 0;
    while (token != NULL) {
      strcpy(genesetGenes[genesetLength], token);
      genesetLength++;
      token = strtok(NULL, "\t");
    }

    printf("%s\n", pathwayName);

    // -------------------------------------------------------------------------
    // Get rid of trailing carriage return on last gene name
    // no longer needed because strok with pathwayName eliminates newline
    // character genesetgenes[genesetlength -
    // 1][strlen(genesetgenes[genesetlength - 1]) - 1] = '\0';

    printf("%d genes in geneset\n", genesetLength);

    // accounts for any missing/extra genes
    int indexPos = 0;
    for (i = 0; i < genesetLength; i++) {
      int flagFound = 0;
      genesetIndexIntoData[i] = -1;
      for (j = 0; j < numGenes; j++) {
        if (!strcmp(geneNames[j], genesetGenes[i])) {
          flagFound = 1;
          // fill genesetgenes only with genes that are being evaluated
          strcpy(genesetGenes[indexPos], geneNames[j]);
          genesetIndexIntoData[indexPos] = j;
          break;
        }
      }

      // printf ("Gene %d index: %d %d %s
      // %s\n",i,genesetindexintodata[i],j,genenames[genesetindexintodata[i]],genesetgenes[i]);
      if (flagFound) {
        indexPos++;
      }
    }
    transferData1 = (int *)malloc(genesetLength * numClass1 * sizeof(int));
    transferData2 = (int *)malloc(genesetLength * numClass2 * sizeof(int));

    // delete genes that shouldn't be in gene list
    for (int k = indexPos; k < genesetLength; k++) {
      genesetGenes[k][0] = '\0';
    }

    // adjust # of genes
    genesetLength = indexPos;
    printf("Adjusted genes : %d\n", genesetLength);

    // prior knowledge load data into binary
    // matrix-------------------------------------------------------------
    int *priorMatrix =
        (int *)calloc(genesetLength * genesetLength,
                      sizeof(int)); // array to hold prior knowledge matrix

    // look into \PRIORS folder
    char directory[300];
    ;                       // directory for prior files
    strcpy(directory, DIR); // load folder path depending on if unix or windows
    char fileName[1000];    // name of prior file
    strcpy(fileName, pathwayName);
    strcat(fileName,
           ".prior"); // take pathwayname and add .prior to get file path
    strcat(directory, fileName);
    if (strstr(directory, "\r") != NULL) {
      printf("File problem! Uses window endings!\n");
    }

    FILE *priorFile = fopen(directory, "r"); // open prior knowledge file
    printf("file : %s\n", directory);
    int priorFlag = 1; // 1 = files found 0 = no file found
    if (priorFile == NULL) {
      printf("No prior file exists. Computing without prior knowledge\n");
      priorFlag = 0;
    }

    // fill prior Matrix
    char priorBuffer[100];
    while (priorFlag && fgets(priorBuffer, sizeof(priorBuffer), priorFile)) {
      char *tok = strtok(priorBuffer, "\t");
      // printf("gene1 : %s\n", tok);
      int insideFlag = 0;
      int row = -1, col = -1;
      for (int k = 0; k < genesetLength; k++) {
        if (strcmp(genesetGenes[k], tok) == 0) {
          insideFlag = 1;
          row = k;
          break;
        }
      }

      if (insideFlag == 0)
        continue;
      tok = strtok(NULL, "\t");
      // printf("relationship : %s\n", tok);
      if (strcmp(tok, "neighbor-of") == 0) {
        continue;
      }
      tok = strtok(NULL, "\t");
      tok[strlen(tok) - 1] = '\0';
      // printf("gene2 : %s\n", tok);
      insideFlag = 0;
      for (int k = 0; k < genesetLength; k++) {
        if (strcmp(genesetGenes[k], tok) == 0) {
          insideFlag = 1;
          col = k;
          break;
        }
      }
      if (insideFlag == 0)
        continue;
      assert(row > -1 && col > -1 && row < genesetLength &&
             col < genesetLength);
      *(priorMatrix + row * genesetLength + col) = 1;
      *(priorMatrix + col * genesetLength + row) = 1;
    }

    if (priorFlag == 1) // only try closing file if it was open to begin with
    {
      fclose(priorFile);
    }

    // begin permutation loop
    int n;
    int x;

    // stores js values across permutations for p value calcs
    double *jsVals = (double *)malloc(sizeof(double) * perms);

    // used to print network/bdeu score files
    int first_unisum;
    int first_scaler;
    int *first_uniNodes;
    int *first_uniEdges;
    int *first_uniEpn;
    double *first_lval1 = NULL;
    int first_numEdges;
    int *uniqueNetIds = NULL;

    // used to calculate edgeList without parent limit after permuatations
    // finished - stores 1st permutation data
    int *edgeListData1 = NULL;
    int *edgeListData2 = NULL;

    int *initialSearcher = NULL;

    // number range of random numbers needed [0,mems)
    int mems = numSamples;

    float totalTime;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // numClass1 = n_samples_C1, numClass2 = n_samples_C2
    if (prop_resampling > 0) {
      n_resampled_C1 = get_proportional_resampling_size(n_resamplings, numClass1);
      n_resampled_C2 = get_proportional_resampling_size(n_resamplings, numClass2);      
    } else {
      n_resampled_C1 = get_leave_one_out_resampling_size(numClass1);
      n_resampled_C2 = get_leave_one_out_resampling_size(numClass2);
    }

    resampled_indices_C1 = (short *)malloc(sizeof(short) * n_resampled_C1);
    resampled_indices_C2 = (short *)malloc(sizeof(short) * n_resampled_C2);


    printf("Permutations begin:\n");
    for (int permNum = 0; permNum < perms; permNum++) {
      n = 0;

      int *randNums = (int *)malloc(sizeof(int) * numSamples);
      for (int c = 0; c < mems; c++) {
        randNums[c] = rand() % mems;
      }

      while (n < mems) {
        int r = rand() % mems;

        for (x = 0; x < n; x++) {
          if (randNums[x] == r) {
            break;
          }
        }

        if (x == n) {
          randNums[n++] = r;
        }
      }

      // after first permutation scramble samplings
      if (permNum > 0) {
        for (int counter = 0; counter < numClass1; counter++) {
          assert(counter < numSamples);
          classIDs[randNums[counter]] = 0;
        }
        for (int counter = numClass1; counter < numSamples; counter++) {
          assert(counter < numSamples);
          classIDs[randNums[counter]] = 1;
        }
      }

      free(randNums);
      randNums = NULL;

      // sort data into class1 and class 2
      int index = 0;
      for (i = 0; i < genesetLength; i++) {
        if (genesetIndexIntoData[i] == -1) {
          i++;
        }

        int jindex1 = 0;
        int jindex2 = 0;
        for (j = 0; j < numSamples; j++) {
          if (classIDs[j] == 0) {
            assert(index * numClass1 + jindex1 <
                   numClass1 * genesetLength); // trDnsferdata
            assert(genesetIndexIntoData[i] * numSamples + j <
                   numGenes * numSamples); // data
            transferData1[index * numClass1 + jindex1] =
                data[genesetIndexIntoData[i] * numSamples + j];
            jindex1++;
          }
          if (classIDs[j] == 1) {
            assert(index * numClass2 + jindex2 <
                   numClass2 * genesetLength); // Dransferdata
            assert(genesetIndexIntoData[i] * numSamples + j <
                   numGenes * numSamples); // data
            transferData2[index * numClass2 + jindex2] =
                data[genesetIndexIntoData[i] * numSamples + j];
            jindex2++;
          }
        }
        index++;
      }

      int genes = genesetLength;

      // start timing- Tomas
      hipEventRecord(begin, 0);

      int samples = numClass1;
      int samples2 = numClass2;

      int c = (((genes * genes) - genes) / 2);

      int scaler = (samples + 1);
      int scaler2 = (samples2 + 1);
      int scalerSum = scaler + scaler2;
      int num_sample_blocks;
      int sample_num1, sample_num2;

      int *spacer1; // dyn
      spacer1 = (int *)malloc(c * sizeof(int));
      int *ff1; // dyn
      int *searcher;
      ff1 = (int *)malloc(c * sizeof(int));
      searcher = (int *)malloc(genes * sizeof(int));

      // determines diaganol representation of data matrix
      searcher[0] = 0;
      int position = 0;
      for (int row = 1; row < genes; row++) {
        for (int col = 0; col < row; col++) {
          assert(position < c);
          spacer1[position] = row;
          // printf("spacer1[%d] : %d\n", position, spacer1[position]);
          ff1[position] = col;
          position++;
        }
        if (row > 0) {
          assert(row < genes);
          searcher[row] = searcher[row - 1] + row;
        }
      }

      // if first permutation store spacr, ff, searcher to use in edgeList calcs
      // after permutations
      if (permNum == 0) {
        // initialSpacr = (int *)malloc(sizeof(int) * c);
        // initialFF = (int *)malloc(sizeof(int) * c);
        initialSearcher = (int *)malloc(sizeof(int) * genes);
        // memcpy(initialSpacr, spacer1, sizeof(int) * c);
        // memcpy(initialFF, ff1, sizeof(int) * c);
        memcpy(initialSearcher, searcher, sizeof(int) * genes);
      }

      // start cuda time
      hipEvent_t start, stop;
      float time;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      /// cuda launch
      /// 1***************************************************************************
      // holds edge data in binary format
      // int onesSize = sizeof(double) * c * scalerSum;
      int *edgesPN;
      edgesPN = (int *)malloc(sizeof(int) * (scalerSum + 1));

      // device copies for out23 and edgesPN
      int *dout23;
      int *dedgesPN;

      // device copies
      int *dtriA, *ddofout, *dtriAb, *dppn, *dstf;
      int *dff, *dspacr;
      // double *d_ones;
      int *dpriorMatrix;

      // mem sizes required
      int size2 = c * ((samples2 + 1) + (samples + 1)) * sizeof(int);
      // int size3 = c*((samples2 + 1) + (samples + 1))*sizeof(double);
      int dppnLength = genesetLength * 2;
      ////space alloc for device
      HANDLE_ERROR(
          hipMalloc((void **)&dtriA, genesetLength * samples * sizeof(int)));
      HANDLE_ERROR(
          hipMalloc((void **)&dtriAb, genesetLength * samples2 * sizeof(int)));
      HANDLE_ERROR(hipMalloc((void **)&dppn, genesetLength * 2 * sizeof(int)));
      HANDLE_ERROR(
          hipMalloc((void **)&dstf, genesetLength * 2 * 3 * sizeof(int)));
      HANDLE_ERROR(hipMalloc((void **)&ddofout, size2));
      HANDLE_ERROR(hipMalloc((void **)&dff, c * sizeof(int)));
      HANDLE_ERROR(hipMalloc((void **)&dspacr, c * sizeof(int)));
      // hipMalloc((void **)&d_ones, onesSize);
      HANDLE_ERROR(hipMalloc((void **)&dout23, sizeof(int) * c * scalerSum));
      HANDLE_ERROR(
          hipMalloc((void **)&dedgesPN, sizeof(int) * (scalerSum + 1)));

      HANDLE_ERROR(hipMalloc((void **)&dpriorMatrix,
                              sizeof(int) * genesetLength * genesetLength));

      // copy into device
      assert(genes * samples * sizeof(int) ==
             genesetLength * numClass1 * sizeof(int));
      assert(genes * samples2 * sizeof(int) ==
             genesetLength * numClass2 * sizeof(int));
      HANDLE_ERROR(hipMemcpy(dtriA, transferData1,
                              genes * samples * sizeof(int),
                              hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(dtriAb, transferData2,
                              genes * samples2 * sizeof(int),
                              hipMemcpyHostToDevice));
      HANDLE_ERROR(
          hipMemcpy(dff, ff1, c * sizeof(int), hipMemcpyHostToDevice));
      HANDLE_ERROR(
          hipMemcpy(dspacr, spacer1, c * sizeof(int), hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(dpriorMatrix, priorMatrix,
                              genesetLength * genesetLength * sizeof(int),
                              hipMemcpyHostToDevice));

      // no longer used once copied to GPU
      free(spacer1);
      spacer1 = NULL;
      free(ff1);
      ff1 = NULL;

      // deploy
      int milliSecondsElapsed1 = getMilliSpan(start1);
      int start2 = getMilliCount();
      //int sampleSum = samples + samples2 + 2;
      // printf("samples : %d\n", samples);

      // run no states in separate kernel to avoid threading
      // noStates_kernel <<<genes * 2, 1 >>>(genes, samples, samples2, dtriA,
      // dtriAb, dppn, dstf);



      // resampling indices
      //
      if (prop_resampling > 0) {
        construct_proportional_resampling_indices(n_resamplings, samples, prop_resampling, resampled_indices_C1);
        construct_proportional_resampling_indices(n_resamplings, samples2, prop_resampling, resampled_indices_C2);        
        num_sample_blocks=n_resamplings*2+2;
        sample_num1=n_resamplings;
        sample_num2=n_resamplings;
      } else {
        printf("Leave one out resampling\n");
        construct_leave_one_out_resampling_indices(samples, resampled_indices_C1);
        construct_leave_one_out_resampling_indices(samples2, resampled_indices_C2);        
        num_sample_blocks=scalerSum;
        sample_num1=samples;
        sample_num2=samples2;
      }

      /*for (i=0;i<n_resampled_C1;i++) {
        printf("C1 INDEX %d: %d\n",i,resampled_indices_C1[i]);
      }
      for (i=0;i<n_resampled_C2;i++) {
        printf("C2 INDEX %d: %d\n",i,resampled_indices_C2[i]);
      }*/

      short *d_resampling_C1, *d_resampling_C2;
      
      // int size_resamp = n_resamplings*sizeof(short);
      HANDLE_ERROR(hipMalloc(&d_resampling_C1, sizeof(short)*n_resampled_C1));
      HANDLE_ERROR(hipMalloc(&d_resampling_C2, sizeof(short)*n_resampled_C2));
      HANDLE_ERROR(hipMemcpy(d_resampling_C1, resampled_indices_C1, sizeof(short)*n_resampled_C1, hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(d_resampling_C2, resampled_indices_C2, sizeof(short)*n_resampled_C2, hipMemcpyHostToDevice));


      hipEventRecord(start, 0);
      // printf("c = %d\n", c);
      printf("SAMPLE1 = %d SAMPLES2 = %d\n", sample_num1, sample_num2);

#if __DEBUG_EDDY__
        printf ("BEFORE RUN2\n");
#endif  /* __DEBUG_EDDY__ */

      if (c < MAX_THREADS) {

        // run2<<<sampleSum, c, genes * genes * sizeof(int)>>>(
        //     genes, samples, samples2, dtriA, dtriAb, dspacr, dff, ddofout, dppn,
        //     dstf, dout23, c, dpriorMatrix, alphaEdgePrior, alphaEdge,
        //     flag_pAdjust);

        printf ("DE_RESAMP %d\n",num_sample_blocks);
        determineEdges_resampled<<<num_sample_blocks, c>>>(d_resampling_C1, d_resampling_C2,
                                 genes, samples, samples2, sample_num1, dtriA, dtriAb, dpriorMatrix, 
                                 alphaEdgePrior, alphaEdge, flag_pAdjust,
                                 dppn, dstf, dspacr, dff, ddofout, c, dout23);
      } else {
        int BPN = ceil((c * 1.0) / MAX_THREADS);
        int TPB = ceil((c * 1.0) / BPN);

 	      printf("C (g*G/2) = %d\n",c);
        printf("launching with %d blocks per network and %d threads per block\n", BPN, TPB);
        determineEdges_resampled_scalable<<<num_sample_blocks * BPN, TPB>>>(
            d_resampling_C1, d_resampling_C2,
            genes, samples, samples2, sample_num1, dtriA, dtriAb, dpriorMatrix, 
            alphaEdgePrior, alphaEdge, flag_pAdjust, 
            dppn, dstf, dspacr, dff, ddofout, c, dout23,
            BPN, TPB);
        //run2Scalable<<<sampleSum * BPN, TPB>>>(
        //    genes, samples, samples2, dtriA, dtriAb, dspacr, dff, ddofout, dppn,
        //    dstf, dout23, c, dpriorMatrix, alphaEdgePrior, alphaEdge,
        //    flag_pAdjust, BPN, TPB);
        printf("run2Scalable completed\n");
      }

#if __DEBUG_EDDY__
        printf ("AFTER RUN2\n");
#endif /* __DEBUG_EDDY__ */

      // test ppn/stf
      /*int *tempPpn = (int *)malloc(sizeof(int) * 2 * genesetlength);
         int *tempStf = (int *)malloc(sizeof(int) * 2 * 3 * genesetlength);
         hipMemcpy(tempPpn, dppn, sizeof(int) * 2 * genesetlength,
         hipMemcpyDeviceToHost); hipMemcpy(tempStf, dstf, sizeof(int) * 2 * 3
         * genesetlength, hipMemcpyDeviceToHost); for (int i = 0; i < 2 *
         genesetlength; i++)
         {
         printf("ppn[%d] : %d\n", i, tempPpn[i]);
         }
         for (int i = 0; i < 2 * 3 * genesetlength; i++)
         {
         printf("stf[%d] : %d\n", i, tempStf[i]);
         } */



      // printf("run2 finished\n");
      hipError_t errSync = hipGetLastError();
      if (errSync != hipSuccess) {
        printf("%s\n", hipGetErrorString(errSync));
      }
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);
      // printf("Run 2 Time : %f\n", time);

      if (permNum == 0) {
        // holds post run2 edge data for edge list calculations after
        // permutations
        edgeListData1 = (int *)malloc(sizeof(int) * c);
        edgeListData2 = (int *)malloc(sizeof(int) * c);

        // host array to transfer output of run2 to edgeListData1/edgeListData2
        int *tempOut23 = (int *)malloc(sizeof(int) * c * num_sample_blocks);

        // printf ("LINE NUMBER 811\n");
        // copy binary data back to CPU
        HANDLE_ERROR(hipMemcpy(tempOut23, dout23, sizeof(int) * c * num_sample_blocks,
                                hipMemcpyDeviceToHost));
        printf ("LINE NUMBER 815\n");

        // first network in first class - no samples left out
        for (int i = 0; i < c; i++) {
          // load 1st network from class 1
          edgeListData1[i] = tempOut23[i];
        }
        int count = 0;
        // last network in 2nd class - no samples left out
        for (int i = (num_sample_blocks - 1) * c; i < (num_sample_blocks)*c; i++) {
          edgeListData2[count++] = tempOut23[i];
        }

        ////copy data for the first network in the first class
        // int *ptr1 = &tempOut23[0];
        // memcpy(edgeListData1, ptr1, sizeof(int) * c);
        ////copy data for the first network in the second class
        // printf("2nd memcpy starting point : %d\n", scaler * c);
        ////int *ptr2 = &tempOut23[scaler * c];
        // int *ptr2 = &tempOut23[(scaler) * c];
        // memcpy(edgeListData2, ptr2, sizeof(int) * c);
        // ptr1 = NULL;
        // ptr2 = NULL;

        free(tempOut23);
        tempOut23 = NULL;
      }

      int milliSecondsElapsed2 = getMilliSpan(start2);
      int start3 = getMilliCount();

      // device copy
      int *dsrchAry, *tempEdgesSums;
      HANDLE_ERROR(hipMalloc((void **)&dsrchAry, genes * sizeof(int)));
      HANDLE_ERROR(hipMemcpy(dsrchAry, searcher, genes * sizeof(int),
                              hipMemcpyHostToDevice));
      tempEdgesSums = (int *)calloc(num_sample_blocks + 1, sizeof(int));

      free(searcher);
      searcher = NULL;

      hipEvent_t PN_start, PN_stop;
      hipEventCreate(&PN_start);
      hipEventCreate(&PN_stop);
      hipEventRecord(PN_start, 0);
      float PN_time;

      // edgePerNetworkKernel << < sampleSum + 1, c, (c * sizeof(int)) >>
      // >(dout23, dedgesPN, dsrchAry, genes, MAX_PARENTS, c);
      printf("BEFORE EPN KERNEL\n");
      edgePerNetworkKernel<<<num_sample_blocks, 1>>>(dout23, dedgesPN, dsrchAry,
                                                 genes, MAX_PARENTS, c);
      HANDLE_ERROR(hipDeviceSynchronize());
      printf("edgesPerNetworkKernel finished\n");
      hipEventRecord(PN_stop, 0);
      // HANDLE_ERROR(hipMemcpy(edgesPN, dedgesPN, sizeof(int) * (scalerSum +
      // 1), hipMemcpyDeviceToHost));

      // copy edge sums over to CPU to calculate prefix sum for edgesPN
      HANDLE_ERROR(hipMemcpy(tempEdgesSums, dedgesPN,
                              sizeof(int) * (num_sample_blocks + 1),
                              hipMemcpyDeviceToHost));

      edgesPN[0] = 0;
      for (int i = 1; i < num_sample_blocks + 1; i++) {
        edgesPN[i] =
            edgesPN[i - 1] + tempEdgesSums[i - 1]; // prefix sum calculation
      }
      // get rid of this temp array
      free(tempEdgesSums);
      tempEdgesSums = NULL;
      // edgesPN on the CPU is now fixed but dedgesPN is used later- copy
      // edgesPN results back to GPU memory
      HANDLE_ERROR(hipMemcpy(dedgesPN, edgesPN, sizeof(int) * (num_sample_blocks + 1),
                              hipMemcpyHostToDevice));

     /* 
         for (int i = 0; i < num_sample_blocks + 1; i++)
         {
         printf("edgesPN[%d] : %d\n", i, edgesPN[i]);
         }*/
       
      // exit(EXIT_FAILURE);

      errSync = hipGetLastError();
      if (errSync != hipSuccess) {
        printf("%s\n", hipGetErrorString(errSync));
      }

      /*for (int i = 0; i < scalerSum + 1; i++)
         {
         printf("edgesPN[%d] : %d\n", i, edgesPN[i]);
         } */
      // hipEventRecord(PN_stop, 0);
      hipEventSynchronize(PN_stop);
      hipEventElapsedTime(&PN_time, PN_start, PN_stop);
      // printf("edgesPerNetworkKernel time : %f\n", PN_time);
      // hipFree(d_ones);
      HANDLE_ERROR(hipFree(dpriorMatrix));
      dpriorMatrix = NULL;
      HANDLE_ERROR(hipFree(ddofout));
      ddofout = NULL;
      HANDLE_ERROR(hipFree(dff));
      dff = NULL;
      HANDLE_ERROR(hipFree(dspacr));
      dspacr = NULL; // hipFree(dtriA); hipFree(dtriAb);-used later in run4
      /***********************************************************************************************************************************************************/
      // total number of edges
      int numEdges = edgesPN[num_sample_blocks];
      printf("NUMEDGES = %d\n",numEdges);

      // int N = c;
      // int M = genesetlength - 1;
      // int size1 = sizeof(int)*N*(scalerSum);
      // int size222 = sizeof(double)*N*(scalerSum);
      //*****************************************************************************************
      // run22 launch- create parent graphs
      int noNodes = genesetLength;
      // host copies
      int *pNodes, *pEdges;

      // dev copies
      int *dpEdges, *dpNodes;

      // mem reqs
      int nodeSize = sizeof(int) * (noNodes * (num_sample_blocks));
      int edgeSize = sizeof(int) * numEdges;

      // space alloc for device
      HANDLE_ERROR(hipMalloc((void **)&dpEdges, edgeSize));
      HANDLE_ERROR(hipMalloc((void **)&dpNodes, nodeSize));

      // space alloc for host
      pNodes = (int *)malloc(nodeSize);
      pEdges = (int *)malloc(edgeSize);
      // FILE *edgePNFile = fopen("edgePN2.txt", "w");
      // for(int i = 0; i < scalerSum + 1; i++)
      //{
      //      fprintf(edgePNFile, "edgesPN[%d] : %d\n", i, edgesPN[i]);
      //}
      // fclose(edgePNFile);
      printf("run22 started\n");
      run22<<<num_sample_blocks, noNodes>>>(c, dedgesPN, dout23, dpNodes, noNodes,
                                    numEdges, dsrchAry, dpEdges, MAX_PARENTS);
      printf("run22 finished\n");
      printf("NODE SIZE=%d\n",nodeSize);

      HANDLE_ERROR(
          hipMemcpy(pNodes, dpNodes, nodeSize, hipMemcpyDeviceToHost));
      HANDLE_ERROR(
          hipMemcpy(pEdges, dpEdges, edgeSize, hipMemcpyDeviceToHost));

/*      for (int i = 0; i < nodeSize / sizeof(int); i++)
         {
         if (i > edgeSize / sizeof(int))
         {
         printf("nodes[%d] : %d\n", i, pNodes[i]);
         }
         else
         {
         printf("nodes[%d] : %d edges[%d] : %d\n", i, pNodes[i], i, pEdges[i]);
         }
         } */
         i=1189;
         printf("nodes[%d] : %d edges[%d] : %d\n", i, pNodes[i], i, pEdges[i]);
         i=1190;
         printf("nodes[%d] : %d edges[%d] : %d\n", i, pNodes[i], i, pEdges[i]);

      /*if (permNum == 0)
         {

         for (int i = 0; i < noNodes; i++)
         {
         printf("pNodes[%d] : %d\n", i, pNodes[i]);
         }
         for (int i = 11 * noNodes; i < (11 * noNodes) + noNodes; i++)
         {
         printf("pNodes[%d] : %d\n", i, pNodes[i]);
         }
         } */

      // ensure parent limit
      checkParentLimit(num_sample_blocks, noNodes, MAX_PARENTS, pNodes,
                       nodeSize / sizeof(int));
      /*for (int i = 0; i < nodeSize / sizeof(int); i++)
         {
         if (i > edgeSize / sizeof(int))
         {
         printf("pNodes[%d] : %d\n", i, pNodes[i]);
         }
         else
         {
         printf("pNodes[%d] : %d\t pEdges[%d] : %d\n", i, pNodes[i], i,
         pEdges[i]);
         }
         } */
      /*FILE *outputFile = fopen("NodesEdges2.txt", "w");
         for(int i = 0; i < nodeSize / sizeof(int); i++)
         {
         fprintf(outputFile, "pNodes[%d] : %d\n", i, pNodes[i]);
         }
         for(int i = 0; i < edgeSize / sizeof(int); i++)
         {
         fprintf(outputFile, "pEdges[%d] : %d\n", i, pEdges[i]);
         }

         fclose(outputFile); */
      // printf("%d\n", numEdges);

      HANDLE_ERROR(hipFree(dsrchAry));
      dsrchAry = NULL;
      HANDLE_ERROR(hipFree(dout23));
      dout23 = NULL;
      // end run
      // 22**********************************************************************************************/

      // start processs to identify unique networks
      //int scalerCombo = (scalerSum * scalerSum - scalerSum) / 2;
      int scalerCombo = (num_sample_blocks * num_sample_blocks - num_sample_blocks) / 2;
      // host
      int *scalerTest; // compare value
      int *shrunk;
      int *shrunkPlc; // compare to
      scalerTest = (int *)malloc(sizeof(int) * scalerCombo);
      shrunk = (int *)malloc(sizeof(int) * scalerCombo);
      shrunkPlc = (int *)malloc(sizeof(int) * scalerCombo);

      // see line 132 for more info
      idPrep(num_sample_blocks, scalerCombo, scalerTest, shrunkPlc);

      // dev copies
      // launch for run 25
      // *****************************************************************************************
      int *dshrunk;
      int *dscalerTest;
      int *dshnkplc;
      HANDLE_ERROR(hipMalloc((void **)&dshrunk, sizeof(int) * scalerCombo));
      HANDLE_ERROR(
          hipMalloc((void **)&dscalerTest, sizeof(int) * scalerCombo));
      HANDLE_ERROR(hipMalloc((void **)&dshnkplc, sizeof(int) * scalerCombo));

      // cp into device
      HANDLE_ERROR(hipMemcpy(dscalerTest, scalerTest,
                              sizeof(int) * scalerCombo,
                              hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(dshnkplc, shrunkPlc, sizeof(int) * scalerCombo,
                              hipMemcpyHostToDevice));
      //************************************************************************************************
      //(int scaler,int noEdges, int gLength,int scalerCombo, int *dedgesPN, int
      //*dNodes, int *dedgeAry, int *shrunk)
      //

      // printf("%/ max: %d   ", maxBlocks*maxThreads);
      // printf("\n");
      run25<<<(scalerCombo / (maxThreads - 1)) + 1, maxThreads - 1>>>(
          sample_num1 + 1, num_sample_blocks, numEdges, genesetLength, scalerCombo,
          dedgesPN, dpNodes, dpEdges, dshrunk, dscalerTest, dshnkplc);
      // printf("run25 finished\n");
      //*********************************************************************************************
      HANDLE_ERROR(hipMemcpy(shrunk, dshrunk, sizeof(int) * scalerCombo,
                              hipMemcpyDeviceToHost));
      //*************************test****************************************
      hipFree(dshrunk);
      dshrunk = NULL;
      hipFree(dscalerTest);
      dscalerTest = NULL;
      hipFree(dshnkplc);
      dshnkplc = NULL;
      hipFree(dedgesPN);
      dedgesPN = NULL;
      hipFree(dpEdges);
      dpEdges = NULL;
      hipFree(dpNodes);
      dpNodes = NULL;
      free(shrunkPlc);
      shrunkPlc = NULL;

      bool *uniqueN, *visted;

      // routine for creatation of unique structures
      uniqueN = (bool *)malloc(sizeof(bool) * num_sample_blocks);

      uniqueN[0] = true;
      visted = (bool *)malloc(sizeof(bool) * num_sample_blocks);
      visted[0] = true;

      for (int p = 0; p < num_sample_blocks; p++) {
        visted[p] = false;
      }
      for (int p = 0; p < scalerCombo; p++) {
        assert(scalerTest[p] < num_sample_blocks);
        if (visted[scalerTest[p]] == true) {
          continue;
        } else {
      HANDLE_ERROR(hipDeviceSynchronize());
          if (shrunk[p] == 0) {
            uniqueN[scalerTest[p]] = false;
            visted[scalerTest[p]] = true;
          } else {
            uniqueN[scalerTest[p]] = true;
          }
        }
      }
      // grab network ids from 1st permutation before unique graphs are
      // identified- used when network file is written
      if (permNum == 0) {
        uniqueNetIds = (int *)malloc(sizeof(int) * num_sample_blocks);
        int counter = 0;
        for (int i = 0; i < num_sample_blocks; i++) {
          if (uniqueN[i]) {
            uniqueNetIds[counter++] = i;
          }
        }
        uniqueNetIds = (int *)realloc(uniqueNetIds, counter * sizeof(int));
      }

      free(scalerTest);
      scalerTest = NULL;
      free(shrunk);
      shrunk = NULL;
      free(visted);
      visted = NULL;

      int unisum = 0;
      int edSum = 0;
      // should it be num_sample_blocks or snum_sample_blocks + 1?
      for (int p = 0; p < num_sample_blocks; p++) {

        if (uniqueN[p] == 1) {
          unisum++;
          if (p == num_sample_blocks - 1) {
            assert(p < num_sample_blocks + 1);
            edSum = edSum + (numEdges - edgesPN[p]);

          } else {
            assert(p < num_sample_blocks + 1);
            edSum = edSum + edgesPN[p + 1] - edgesPN[p];
          }
        }
      }

      if (permNum == 0) {
        printf("Original Number of unique Networks : %d\n", unisum);
      }

      // printf("Number of unique networks : %d\n edSum : %d\n numEdges : %d\n
      // edgesPN : %d\n", unisum, edSum, numEdges, edgesPN[scalerSum]);
      //**********************************restructure all
      //************************************************** printf("edSum %d
      // numEdges %d\n", edSum, numEdges);
      int *pUniNodes, *pUniEdges, *pUniEpn;
      // space alloc
      pUniNodes = (int *)malloc(sizeof(int) * unisum * noNodes);
      pUniEdges = (int *)malloc(sizeof(int) * edSum);
      int uniEpnSize = unisum + 1;
      // pUniEpn = (int *)malloc(sizeof(int)*unisum + 1);
      pUniEpn = (int *)malloc(sizeof(int) * uniEpnSize);
      // printf("size of pUniEpn : %d\n", uniEpnSize);

      structureUnique(unisum, numEdges, sample_num1+1, num_sample_blocks, noNodes, uniqueN,
                      edgesPN, pEdges, pNodes, pUniEdges, pUniNodes, pUniEpn);
      // printf("structureUnique (NOT A KERNEL) finished\n");
      /*for (int i = 0; i < uniEpnSize; i++)
         {
         printf("pUniEpn[%d] : %d\n", i, pUniEpn[i]);
         } */
      free(edgesPN);
      edgesPN = NULL;
      free(pNodes);
      pNodes = NULL;
      free(pEdges);
      pEdges = NULL;
      free(uniqueN);
      uniqueN = NULL;

      // ensure parent limit
      checkParentLimit(unisum, noNodes, MAX_PARENTS, pUniNodes,
                       unisum * noNodes);
      /*for (int i = 0; i < unisum * noNodes; i++)
         {
         if (i > edSum)
         {
         printf("pNodes[%d] : %d\n", i, pUniNodes[i]);
         }
         else
         {
         printf("pNodes[%d] : %d\t pEdges[%d] : %d\n", i, pUniNodes[i], i,
         pUniEdges[i]);
         }
         }

         for (int i = 0; i < edSum; i++)
         {
         printf("pUniEdges[%d] : %d\n", i, pUniEdges[i]);
         }
         printf("%d\n", edSum); */

      if (permNum == 0) {
        // store graph data for network file write after permutations finished
        // first_uniEpn = (int *)malloc(sizeof(int) * unisum);
        first_uniEpn = (int *)malloc(sizeof(int) * uniEpnSize);
        first_uniNodes = (int *)malloc(sizeof(int) * unisum * noNodes);
        first_uniEdges = (int *)malloc(sizeof(int) * edSum);
        memcpy(first_uniEpn, pUniEpn, unisum * sizeof(int));
        memcpy(first_uniNodes, pUniNodes, unisum * noNodes * sizeof(int));
        memcpy(first_uniEdges, pUniEdges, edSum * sizeof(int));
        first_numEdges = edSum;
        first_unisum = unisum;
      }

      scaler = unisum;
      if (permNum == 0) {
        first_scaler = scaler;
      }
      numEdges = edSum;
      int uniNodeSize = sizeof(int) * (noNodes * unisum);
      int uniEdgeSize = sizeof(int) * numEdges;

      // cuda run
      // 4(final)*************************************************************************************
      double *out5;

      // dev copies
      // int *dtri1; int *dtri2;
      double *dout5;
      int *dpEdges2;
      int *dpNodes2;
      int *dNij;
      int *dNijk;
      int *dUniEpn;
      // space alloc dev
      HANDLE_ERROR(hipMalloc((void **)&dpEdges2, uniEdgeSize));
      HANDLE_ERROR(hipMalloc((void **)&dpNodes2, uniNodeSize));
      HANDLE_ERROR(hipMalloc((void **)&dUniEpn, sizeof(int) * unisum));
      // HANDLE_ERROR(hipMalloc((void **)&dUniEpn, sizeof(int)*uniEpnSize));
      HANDLE_ERROR(
          hipMalloc((void **)&dout5, sizeof(double) * noNodes * scaler * 2));
      HANDLE_ERROR(
          hipMalloc((void **)&dNij, sizeof(int) * noNodes * scaler * 54));
      HANDLE_ERROR(
          hipMalloc((void **)&dNijk, sizeof(int) * noNodes * scaler * 162));

      // cp to devp'4

      HANDLE_ERROR(
          hipMemcpy(dpEdges2, pUniEdges, uniEdgeSize, hipMemcpyHostToDevice));
      HANDLE_ERROR(
          hipMemcpy(dpNodes2, pUniNodes, uniNodeSize, hipMemcpyHostToDevice));
      // HANDLE_ERROR(hipMemcpy(dUniEpn, pUniEpn, sizeof(int)*uniEpnSize,
      // hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(dUniEpn, pUniEpn, sizeof(int) * unisum,
                              hipMemcpyHostToDevice));
      free(pUniNodes);
      pUniNodes = NULL;
      free(pUniEdges);
      pUniEdges = NULL;
      free(pUniEpn);
      pUniEpn = NULL;

      hipEvent_t run4Start, run4End;
      hipEventCreate(&run4Start);
      hipEventCreate(&run4End);
      hipEventRecord(run4Start, 0);
      float run4Time;

      printf("run 4 start: %d blocks %d threads\n",scaler*2,noNodes);
      run4<<<scaler * 2, noNodes>>>(scaler, dUniEpn, genesetLength, edSum,
                                    unisum, samples, samples2, dtriA, dtriAb,
                                    dpEdges2, dpNodes2, dppn, dstf, dNij, dNijk,
                                    dout5, dppnLength);
      HANDLE_ERROR(hipDeviceSynchronize());
      printf("run 4 finished\n");
      hipEventRecord(run4End, 0);
      HANDLE_ERROR(hipEventSynchronize(run4End));
      hipEventElapsedTime(&run4Time, run4Start, run4End);
      printf("run 4 time : %f\n", run4Time);

      // space alloc host
      out5 = (double *)malloc(sizeof(double) * noNodes * scaler * 2);

      HANDLE_ERROR(hipMemcpy(out5, dout5,
                              sizeof(double) * noNodes * scaler * 2,
                              hipMemcpyDeviceToHost));

      HANDLE_ERROR(hipFree(dNij));
      dNij = NULL;
      HANDLE_ERROR(hipFree(dNijk));
      dNijk = NULL;
      HANDLE_ERROR(hipFree(dppn));
      dppn = NULL;
      HANDLE_ERROR(hipFree(dstf));
      dstf = NULL;
      HANDLE_ERROR(hipFree(dout5));
      dout5 = NULL;
      HANDLE_ERROR(hipFree(dpEdges2));
      dpEdges2 = NULL;
      HANDLE_ERROR(hipFree(dpNodes2));
      dpNodes = NULL;
      HANDLE_ERROR(hipFree(dUniEpn));
      dUniEpn = NULL;
      HANDLE_ERROR(hipFree(dtriA));
      dtriA = NULL;
      HANDLE_ERROR(hipFree(dtriAb));
      dtriAb = NULL;

      hipError_t last = hipGetLastError();
      if (last != hipSuccess) {
        printf("%s\n", hipGetErrorString(last));
      }
      // int div = 0;
      // end final cuda run
      // ***********************************************************************

      // begin divergence calc
      double *lval1;
      lval1 = (double *)malloc(sizeof(double) * scaler * 2);

      for (int i = 0; i < scaler * 2; i++) {
        lval1[i] = 0.0;
      }

      // compute likelihood of different dataset parsed by 2 iterations
      for (int g = 0; g < 2; g++) {
        int set = 0;
        int place = 0;
        double scoreSum = 0;
        double *likeli1;
        double min = 0;
        double max = 0;
        double inAlpha = 0;
        double probScale = 0;
        double likeliSum = 0;
        double nonInf = 0;
        double *dist;

        double *adjusted;
        double *infFlag;
        double *outq;
        int localoffset;
        outq = out5;
        if (g < 1) {
          localoffset = 0;
        } else {
          localoffset = scaler;
        }

        dist = (double *)malloc(sizeof(double) * scaler);
        likeli1 = (double *)malloc(sizeof(double) * scaler);
        adjusted = (double *)malloc(sizeof(double) * scaler);
        infFlag = (double *)malloc(sizeof(double) * scaler);
        for (int k2 = 0; k2 < scaler; k2++) {
          dist[k2] = 0.0;

          adjusted[k2] = 0.0;
          infFlag[k2] = 0.0;
        }
        for (int i = 0; i < scaler * noNodes; i++) {
          dist[place] += outq[i + localoffset * noNodes];
          set++;
          if (set == noNodes) {
            set = 0;
            place++;
          }
        }

        min = dist[0];
        max = dist[0];
        for (int j3 = 0; j3 < scaler; j3++) {
          //                      printf(" dis %d %f \n", j3, dist[j3]);
          scoreSum += dist[j3];
          if (dist[j3] > max) {

            max = dist[j3];
          }
          if (dist[j3] < min) {

            min = dist[j3];
          }
        }

        inAlpha = -1 * (scoreSum / scaler);
        // printf("\n min-%f max-%f", min, max);
        // printf("inAlpha-%f", inAlpha);
        probScale = (10) / (max - min);

        for (int m = 0; m < scaler; m++) {

          adjusted[m] = (dist[m] + inAlpha) * probScale;
          //                      printf("\n adjusted: %f", adjusted[m]);
          likeli1[m] = exp(adjusted[m]);
          //                      printf("\n likeli: %f", likeli1[m]);
          nonInf += likeli1[m];
          // likeLi[m] = posInf;
          // suppress overflow infinity error
#pragma warning(suppress : 4056)
          if (likeli1[m] >= INFINITY || likeli1[m] <= -INFINITY) {
            infFlag[m] = 1.0;
            likeliSum++;
          }
        }
        free(dist);
        dist = NULL;
        free(adjusted);
        adjusted = NULL;
        //      printf("\n likesum: %f nonInf: %f", likeliSum, nonInf);

        if (likeliSum == 0) {
          for (int meow = 0; meow < scaler; meow++) {
            likeli1[meow] = likeli1[meow] / nonInf;
            lval1[meow + localoffset] = likeli1[meow];
          }

        } else {
          for (int meow = 0; meow < scaler; meow++) {
            likeli1[meow] = infFlag[meow] / likeliSum;
            lval1[meow + localoffset] = likeli1[meow];
          }
        }

        free(likeli1);
        likeli1 = NULL;
        free(infFlag);
        infFlag = NULL;
        outq = NULL;
      }

      if (permNum == 0) {
        first_lval1 = (double *)malloc(sizeof(double) * scaler * 2);
        memcpy(first_lval1, lval1, sizeof(double) * scaler * 2);
      }

      /*******************************************************************************************************************************************/
      double *sea;
      sea = (double *)malloc(sizeof(double) * scaler);

      // scaler unique number of networks
      for (int i = 0; i < scaler; i++) {
        sea[i] = (lval1[i] + lval1[i + scaler]) / 2;
      }

      double js = 0;
      double logger = log(2.0);

      // final score
      js = kool(lval1, sea, 0, scaler) / 2 +
           kool(lval1, sea, scaler, scaler) / 2;
      // printf("\njs: %f\n", js / logger);
      assert(permNum < perms);
      jsVals[permNum] = js / logger;

      // printf("permutation : %d\n", permNum);
      if (isnan(jsVals[0])) {
        printf("jsVal[0] NAN--Breaking permutation loop\n");
        jsVals[0] = -999.0;
        break;
      }

      /****************************time**********************************/
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&time, start, stop);

      /*************************************************/

      // printf("\n Time1 for the kernel: %f ms\n", time);
      // printf("\n\n");

      free(sea);
      sea = NULL;
      free(out5);
      out5 = NULL;
      free(lval1);
      lval1 = NULL;

      if ((permNum > 0) && (permNum % 100 == 0)) {
        // print every 100 permutations
        printf("  Permutation %d finished\n", permNum);
      }

    } //---------------------------------------------------------------------------
    // for loop ends for permutations.
    printf("permutation loop (%d) finished\n", perms);
    // free prior knowledge matrix- no longer needed

    int nanFlag = 0;
    for (int i = 0; i < perms; i++) {
      if (isnan(jsVals[i])) {
        nanFlag = 1;
        break;
      }
    }
    if (jsVals[0] < 0 || nanFlag) {
      // fprintf(results, "%s %s\n", pathwayName, "GARBAGE VALUE-NAN");
      fprintf(fpResults,
              "%s failed with %d genes- one of the js scores was nan\n",
              pathwayName, genesetLength);
      printf("nan value- quiting!\n\n\n");
      continue;
    }

    // count how many js values are larger than initial run
    printf("Original JS : %f\n", jsVals[0]);

    // double p_val = 0.0;
    // printf("number larger : %d\n", largerTally);
    // p_val = largerTally / (perms * 1.0);
    // printf("p = %f\n", p_val);
    double pDDN = 0;

    if (perms > 0) {
      double mu = mean(jsVals, perms);
      double var = variance(mu, jsVals, perms);
      double alpha = (((1 - mu) / var) - (1 / mu)) * pow(mu, 2);
      double beta = alpha * (1 / mu - 1);
      printf("alpha : %f beta : %f\n", alpha, beta);
      // p = 1 - betaCDF(jsVals[0], alpha, beta);
      pDDN = boost::math::ibetac(alpha, beta, jsVals[0]);

      printf("pDDN = %f\n", pDDN);
      fprintf(fpResults, "%s\t%f\t%f\t%d\n", pathwayName, jsVals[0], pDDN,
              genesetLength);
    }

    if (pDDN < alphaDDN) // statistically significant
    {

      char networkFilePath[600];
      strcpy(networkFilePath, pathwayName);
      strcat(networkFilePath, "_Networks.txt");
      char bdeuFilePath[600];
      strcpy(bdeuFilePath, pathwayName);
      strcat(bdeuFilePath, "_BDEU_SCORES.txt");

      writeNetworkFile(networkFilePath, inputFile, classFile, pathwayName,
                       first_unisum, first_uniEpn, genesetGenes, genesetLength,
                       first_uniNodes, first_uniEdges, first_numEdges,
                       uniqueNetIds);
      writeBdeuScores(bdeuFilePath, inputFile, classFile, pathwayName, class1,
                      class2, first_scaler, first_lval1);

      // printf("\nOriginal JS score : %f\n", jsVals[0]);
      // printf("Original number of unique networks : %d\n", first_unisum);

      //-----------------------------------------------------
      // edgeList calcs
      // edgesPerNetworkKernel --> run22 --> output Edge list
      // NOTE: genesetlength is used to represent the number of nodes aka
      // noNodes as used in prior kernel calls printf("Final run\n");

      // 2 networks are being looked at and c = number of different gene
      // combinations
      int c = (((genesetLength * genesetLength) - genesetLength) / 2);
      int numNetworks = 2;

      // host copies
      int *nodes, *edges, *out23;
      int edgesPN[3];

      out23 = (int *)malloc(sizeof(int) * c * numNetworks);

      // dev copies
      int *dout23, *dsrchAry, *dEdgesPN;

      // copy data taken from first run2 permutation
      int *ptr1 = &out23[0];
      int *ptr2 = &out23[c];
      memcpy(ptr1, edgeListData1, sizeof(int) * c);
      memcpy(ptr2, edgeListData2, sizeof(int) * c);
      ptr1 = NULL;
      ptr2 = NULL;

      // allocate device memory and copy
      HANDLE_ERROR(hipMalloc((void **)&dout23, sizeof(int) * c * numNetworks));
      HANDLE_ERROR(
          hipMalloc((void **)&dEdgesPN, sizeof(int) * (numNetworks + 1)));
      HANDLE_ERROR(hipMalloc((void **)&dsrchAry, genesetLength * sizeof(int)));

      HANDLE_ERROR(hipMemcpy(dout23, out23, sizeof(int) * c * numNetworks,
                              hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(dsrchAry, initialSearcher,
                              genesetLength * sizeof(int),
                              hipMemcpyHostToDevice));

      const int PARENTS_LIMIT = INT_MAX;

      // edgePerNetworkKernel << <numNetworks + 1, c, (c * sizeof(int)) >>
      // >(dout23, dEdgesPN, dsrchAry, genesetlength, PARENTS_LIMIT, c);
      edgePerNetworkKernel<<<numNetworks + 1, 1>>>(
          dout23, dEdgesPN, dsrchAry, genesetLength, PARENTS_LIMIT, c);
      // edgePerNetworkKernel << <numNetworks + 1, c, (c * sizeof(int)) +
      // (genesetlength * sizeof(int)) >> >(dout23, dEdgesPN, dsrchAry,
      // genesetlength, PARENTS_LIMIT, c);

      // edgePerNetworkKernel calculates sum of edges for each network - now we
      // need to perform the prefix calc for edgesPN on the CPU
      int tempEdgeSum[numNetworks + 1];
      HANDLE_ERROR(hipMemcpy(tempEdgeSum, dEdgesPN,
                              sizeof(int) * (numNetworks + 1),
                              hipMemcpyDeviceToHost));
      edgesPN[0] = 0;
      // calc prefix sum
      for (int i = 1; i < numNetworks + 1; i++) {
        edgesPN[i] = edgesPN[i - 1] + tempEdgeSum[i - 1];
      } // copy results of prefix sum back to GPU for use in run22
      HANDLE_ERROR(hipMemcpy(dEdgesPN, edgesPN,
                              sizeof(int) * (numNetworks + 1),
                              hipMemcpyHostToDevice));

      /*for (int i = 0; i < 3; i++)
         {
         printf("edgesPN[%d]  : %d\n", i, edgesPN[i]);
         }

         for (int i = 0; i < genesetlength; i++)
         {
         printf("%d : %s\n", i, genesetgenes[i]);
         } */

      // needed to calculate how long to make edge array
      int totalEdges = edgesPN[2];

      nodes = (int *)malloc(sizeof(int) * genesetLength * 2);
      edges = (int *)malloc(sizeof(int) * totalEdges);

      int *dNodes, *dEdges;
      HANDLE_ERROR(
          hipMalloc((void **)&dNodes, sizeof(int) * genesetLength * 2));
      HANDLE_ERROR(hipMalloc((void **)&dEdges, sizeof(int) * totalEdges));

      run22<<<numNetworks, genesetLength>>>(c, dEdgesPN, dout23, dNodes,
                                            genesetLength, totalEdges, dsrchAry,
                                            dEdges, PARENTS_LIMIT);

      HANDLE_ERROR(hipMemcpy(nodes, dNodes, sizeof(int) * 2 * genesetLength,
                              hipMemcpyDeviceToHost));
      HANDLE_ERROR(hipMemcpy(edges, dEdges, sizeof(int) * totalEdges,
                              hipMemcpyDeviceToHost));

      HANDLE_ERROR(hipFree(dout23));
      dout23 = NULL;
      HANDLE_ERROR(hipFree(dsrchAry));
      dsrchAry = NULL;
      HANDLE_ERROR(hipFree(dEdgesPN));
      dEdgesPN = NULL;
      HANDLE_ERROR(hipFree(dNodes));
      dNodes = NULL;
      HANDLE_ERROR(hipFree(dEdges));
      dEdges = NULL;
      free(out23);
      out23 = NULL;

      char edgeListFile[600];
      strcpy(edgeListFile, pathwayName);
      strcat(edgeListFile, "_EdgeList.txt");
      // int networkIds[2] = { 1, 2 };
      // writeNetworkFile(edgeListFile, inputFile, classFile, pathwayName, 2,
      // edgesPN, genesetgenes, genesetlength, nodes, edges, totalEdges,
      // networkIds);
      writeEdgeListFile(edgeListFile, inputFile, classFile, pathwayName,
                        genesetGenes, genesetLength, nodes, edges, edgesPN,
                        priorMatrix, class1, class2);

      hipEventRecord(end, 0);
      hipEventSynchronize(end);
      hipEventElapsedTime(&totalTime, begin, end);

      // printf("Total Run Time : %f\n", totalTime);

      // FILE *timeFile = fopen("Time.txt", "w");
      // fprintf(timeFile, "%f", totalTime);
      // output number of unique networks to check against java scores
      printf("Total run time : %f\n", totalTime);
      // fprintf(timeFile, "%f", totalTime);
      // fclose(timeFile);
      free(nodes);
      nodes = NULL;
      free(edges);
      edges = NULL;
    }

    printf("\nPathway finished.\n\n");
    free(priorMatrix);
    priorMatrix =
        NULL; // free this after writing files b/c needed for writeEdgeListFile

    diff = clock() - cpuTime;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Time taken %d seconds %d milliseconds\n\n\n", msec / 1000,
           msec % 1000);
    //---------------------------------------------------------------------------
    // free variables

    free(resampled_indices_C1);
    free(resampled_indices_C2);

    free(transferData1);
    free(transferData2);
    transferData1 = NULL;
    transferData2 = NULL;

    free(first_lval1);
    free(first_uniNodes);
    free(first_uniEdges);
    free(first_uniEpn);
    free(jsVals);
    free(uniqueNetIds);
    free(edgeListData1);
    free(edgeListData2);
    // free(initialFF);
    free(initialSearcher);
    // free(initialSpacr);
    first_lval1 = NULL;
    first_uniNodes = NULL;
    first_uniEdges = NULL;
    first_uniEpn = NULL;
    jsVals = NULL;
    uniqueNetIds = NULL;
    edgeListData1 = NULL;
    edgeListData2 = NULL;
    // initialFF = NULL;
    initialSearcher = NULL;
    // initialSpacr = NULL;
  }
  fclose(fpGeneSet);
  fclose(fpResults);

  free(data);
  data = NULL;
  // use to make sure all data is recorded and visual
  // profiler works
  // hipDeviceReset();

  return 0;
}
